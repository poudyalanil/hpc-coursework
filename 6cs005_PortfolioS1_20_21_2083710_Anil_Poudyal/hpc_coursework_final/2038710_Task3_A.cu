#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

// Time header
#include <time.h>
 
//Password Cracking using CUDA 

__device__ char* encryptDecrypt(char* tempPassword){

	char * generatedPassword = (char *) malloc(sizeof(char) * 11);

	generatedPassword[0] = tempPassword[0] + 2;
	generatedPassword[1] = tempPassword[0] - 2;
	generatedPassword[2] = tempPassword[0] + 1;
	generatedPassword[3] = tempPassword[1] + 3;
	generatedPassword[4] = tempPassword[1] - 3;
	generatedPassword[5] = tempPassword[1] - 1;
	generatedPassword[6] = tempPassword[2] + 2;
	generatedPassword[7] = tempPassword[2] - 2;
	generatedPassword[8] = tempPassword[3] + 4;
	generatedPassword[9] = tempPassword[3] - 4;
	generatedPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ 
			if(generatedPassword[i] > 122){
				generatedPassword[i] = (generatedPassword[i] - 122) + 97;
			}else if(generatedPassword[i] < 97){
				generatedPassword[i] = (97 - generatedPassword[i]) + 97;
			}
		}else{ 
			if(generatedPassword[i] > 57){
				generatedPassword[i] = (generatedPassword[i] - 57) + 48;
			}else if(generatedPassword[i] < 48){
				generatedPassword[i] = (48 - generatedPassword[i]) + 48;
			}
		}
	}
	return generatedPassword;
}

__global__ void crack(char * alphabet, char * numbers){

char matchedPassword[4];

matchedPassword[0] = alphabet[blockIdx.x];
matchedPassword[1] = alphabet[blockIdx.y];

matchedPassword[2] = numbers[threadIdx.x];
matchedPassword[3] = numbers[threadIdx.y];


char* encryptedPassword = "xtwcvx5171"; //vy33
char* search = encryptDecrypt(matchedPassword);
int iter = 0;
int is_match = 0;
while (*encryptedPassword != '\0' || *search != '\0') {
	if (*encryptedPassword == *search) {
		encryptedPassword++;
		search++;
	} else if ((*encryptedPassword == '\0' && *search != '\0') || (*encryptedPassword != '\0' && *search == '\0') || *encryptedPassword != *search) {
		is_match = 1;
	
		break;
	}
}
if (is_match == 0) {
	printf("Password Found: %c%c%c%c \n", matchedPassword[0],matchedPassword[1],matchedPassword[2],matchedPassword[3]);
}


}

int calculate_time(struct timespec *start, struct timespec *end,
                   long long int *diff)
{
    long long int in_sec = end->tv_sec - start->tv_sec;
    long long int in_nano = end->tv_nsec - start->tv_nsec;
    if (in_nano < 0)
    {
        in_sec--;
        in_nano += 1000000000;
    }
    *diff = in_sec * 1000000000 + in_nano;
    return !(*diff > 0);
}

int main(int argc, char ** argv){

    struct timespec start, end;
    long long int time_used;

	char cpuLetters[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
	char cpuDigits[26] = {'0','1','2','3','4','5','6','7','8','9'};

	char * gpuLetters;
	hipMalloc( (void**) &gpuLetters, sizeof(char) * 26); 
	hipMemcpy(gpuLetters, cpuLetters, sizeof(char) * 26, hipMemcpyHostToDevice);

	char * gpuDigits;
	hipMalloc( (void**) &gpuDigits, sizeof(char) * 26); 
	hipMemcpy(gpuDigits, cpuDigits, sizeof(char) * 26, hipMemcpyHostToDevice);
    
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);
	crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuLetters, gpuDigits );
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  calculate_time(&start, &end, &time_used);

  printf("Time taken: %f seconds OR %lld Nano Seconds\n", (time_used / 1.0e9), (time_used));
  
    
	return 0;
}












