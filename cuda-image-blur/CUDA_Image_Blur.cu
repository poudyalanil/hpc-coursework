#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

int get_time(struct timespec *start, struct timespec *end,
                   long long int *diff)
{
  long long int in_sec = end->tv_sec - start->tv_sec;
  long long int in_nano = end->tv_nsec - start->tv_nsec;
  if (in_nano < 0)
  {
    in_sec--;
    in_nano += 1000000000;
  }
  *diff = in_sec * 1000000000 + in_nano;
  return !(*diff > 0);
}

__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col, unsigned int* width)
{
		char chaudai = *width;
    unsigned int i = (row * chaudai * 4) + (col * 4);
    return image[i];
}

__device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned int* width)
{
		char chaudai = *width;
    unsigned int i = (row * chaudai * 4) + (col * 4) + 1;
    return image[i];
}

__device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned int* width)
{
		char chaudai = *width;
    unsigned int i = (row * chaudai * 4) + (col * 4) + 2;
    return image[i];
}

__device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned int* width)
{
		char chaudai = *width;
    unsigned int i = (row * chaudai * 4) + (col * 4) + 3;
    return image[i];
}

__device__ void setRed(unsigned char *image, unsigned int row, unsigned int col, unsigned char red, unsigned int* width)
{
		char chaudai = *width;
    unsigned int i = (row * chaudai * 4) + (col * 4);
    image[i] = red;
}

__device__ void setGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned char green, unsigned int* width)
{
		char chaudai = *width;
    unsigned int i = (row * chaudai * 4) + (col * 4) + 1;
    image[i] = green;
}

__device__ void setBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned char blue, unsigned int* width)
{
		char chaudai = *width;
    unsigned int i = (row * chaudai * 4) + (col * 4) + 2;
    image[i] = blue;
}

__device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned char alpha, unsigned int * width)
{
		char chaudai = *width;
		unsigned int i = (row * chaudai * 4) + (col * 4) + 3;
		image[i] = alpha;
}


__global__ void blur(unsigned int *height, unsigned int *width, unsigned int *result, unsigned char * newImage, unsigned char * image){
		unsigned redTL, redTC, redTR;
    unsigned redL, redC, redR;
    unsigned redBL, redBC, redBR;
    unsigned newRed;

    unsigned greenTL, greenTC, greenTR;
    unsigned greenL, greenC, greenR;
    unsigned greenBL, greenBC, greenBR;
    unsigned newGreen;

    unsigned blueTL, blueTC, blueTR;
    unsigned blueL, blueC, blueR;
    unsigned blueBL, blueBC, blueBR;
		unsigned newBlue;
	

		// float filter[3][3] = {
		// 		{1.0 / 10, 1.0 / 10, 1.0 / 10},
		// 		{1.0 / 10, 1.0 / 10, 1.0 / 10},
		// 		{1.0 / 10, 1.0 / 10, 1.0 / 10}};


		float filter[3][3] = {
				{1.0 / 16, 2.0 / 16, 1.0/ 16},
				{2.0 / 16, 4.0 / 16, 2.0 / 16},
				{1.0 / 16, 2.0 / 16, 1.0 / 16}};

	for (unsigned int row = 1; row < *height - 1; row++)
    {
        for (unsigned int col = 1; col < *width - 1; col++)
        {
            setGreen(newImage, row, col, getGreen(image, row, col, width), width);
            setBlue(newImage, row, col, getBlue(image, row, col, width), width);
            setAlpha(newImage, row, col, 255, width);

            redTL = getRed(image, row - 1, col - 1, width);
            redTC = getRed(image, row - 1, col, width);
            redTR = getRed(image, row - 1, col + 1, width);

            redL = getRed(image, row, col - 1, width);
            redC = getRed(image, row, col, width);
            redR = getRed(image, row, col + 1, width);

            redBL = getRed(image, row + 1, col - 1, width);
            redBC = getRed(image, row + 1, col, width);
            redBR = getRed(image, row + 1, col + 1, width);

            newRed = redTL * filter[0][0] + redTC * filter[0][1] + redTR * filter[0][2] + redL * filter[1][0] + redC * filter[1][1] + redR * filter[1][2] + redBL * filter[2][0] + redBC * filter[2][1] + redBR * filter[2][2];

            setRed(newImage, row, col, newRed, width);

            greenTL = getGreen(image, row - 1, col - 1,width);
            greenTC = getGreen(image, row - 1, col,width);
            greenTR = getGreen(image, row - 1, col + 1,width);

            greenL = getGreen(image, row, col - 1,width);
            greenC = getGreen(image, row, col,width);
            greenR = getGreen(image, row, col + 1,width);

            greenBL = getGreen(image, row + 1, col - 1, width);
            greenBC = getGreen(image, row + 1, col, width);
            greenBR = getGreen(image, row + 1, col + 1,width);

            newGreen = greenTL * filter[0][0] + greenTC * filter[0][1] + greenTR * filter[0][2] + greenL * filter[1][0] + greenC * filter[1][1] + greenR * filter[1][2] + greenBL * filter[2][0] + greenBC * filter[2][1] + greenBR * filter[2][2];

            setGreen(newImage, row, col, newGreen, width);

            blueTL = getBlue(image, row - 1, col - 1, width);
            blueTC = getBlue(image, row - 1, col, width);
            blueTR = getBlue(image, row - 1, col + 1, width);

            blueL = getBlue(image, row, col - 1, width);
            blueC = getBlue(image, row, col, width);
            blueR = getBlue(image, row, col + 1, width);

            blueBL = getBlue(image, row + 1, col - 1, width);
            blueBC = getBlue(image, row + 1, col, width);
            blueBR = getBlue(image, row + 1, col + 1, width);
            newBlue = blueTL * filter[0][0] + blueTC * filter[0][1] + blueTR * filter[0][2] + blueL * filter[1][0] + blueC * filter[1][1] + blueR * filter[1][2] + blueBL * filter[2][0] + blueBC * filter[2][1] + blueBR * filter[2][2];
            setBlue(newImage, row, col, newBlue, width);
        }
    }
}

int main(void){
	// haleko
	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width, height;
	const char* filename = "image.png";
	const char* newFileName = "generated.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// haleko end

	unsigned int *gpuWidth, *gpuHeight, *gpuResult;

	hipMalloc(&gpuWidth, sizeof(unsigned int));
	hipMalloc(&gpuHeight, sizeof(unsigned int));
	hipMalloc(&gpuResult, sizeof(unsigned int));

	// width = 2;
	// height = 10;


	hipMemcpy(gpuWidth, &width, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(gpuHeight, &height, sizeof(unsigned int), hipMemcpyHostToDevice);
  
  struct timespec start, end;
  long long int elasped_time;

  clock_gettime(CLOCK_MONOTONIC_RAW, &start);
	blur<<<height, width>>>(gpuHeight, gpuWidth, gpuResult, d_out, d_in);
	// hipMemcpy(&result, gpuResult, sizeof(int), hipMemcpyDeviceToHost);
	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
  clock_gettime(CLOCK_MONOTONIC_RAW, &end);


  get_time(&start, &end, &elasped_time);


	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}
  printf("elasped time  %f s or %lld ns\n", (elasped_time / 1.0e9), (elasped_time));
  
	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
