
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

// #include "lodepng.h"

__global__ void square(int *height, int *width, int *result){
	*result = *height * *width;
}

int main(void){
	int width, height, result;
	int *gpuWidth, *gpuHeight, *gpuResult;

	hipMalloc(&gpuWidth, sizeof(int));
	hipMalloc(&gpuHeight, sizeof(int));
	hipMalloc(&gpuResult, sizeof(int));

	width = 2;
	height = 10;


	hipMemcpy(gpuWidth, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpuHeight, &height, sizeof(int), hipMemcpyHostToDevice);
	
	square<<<height, width>>>(gpuHeight, gpuWidth, gpuResult);
	hipMemcpy(&result, gpuResult, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d\n", result);
	
	return 0;
}
