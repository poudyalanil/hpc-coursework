#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

//compile with c++ lodepng file

//nvcc CudaNegative.cu lodepng.cpp

__global__ void square(unsigned char * gpu_imageOutput, unsigned char * gpu_imageInput){

	int r;
	int g;
	int b;
	int t;
	
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	int pixel = idx*4;
		r = gpu_imageInput[pixel];
		g = gpu_imageInput[1+pixel];
		b = gpu_imageInput[2+pixel];
		t = gpu_imageInput[3+pixel];

		gpu_imageOuput[pixel] = 255-r;
		gpu_imageOuput[1+pixel] = 255-g;
		gpu_imageOuput[2+pixel] = 255-b;
		gpu_imageOuput[3+pixel] = t;
}

int main(int argc, char **argv){

	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "4x4.png";
	const char* newFileName = "generated.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	square<<<height, width>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}

	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
