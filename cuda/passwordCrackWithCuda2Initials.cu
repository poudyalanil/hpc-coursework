#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime_api.h>

#include <time.h>

__device__ int is_a_match(char * attempt) {
  char password1[] = "AP25";
  char password2[] = "AN52";
  char password3[] = "RA25";
  char password4[] = "RC80";

  char * a = attempt;
  char * b = attempt;
  char * c = attempt;
  char * d = attempt;
  char * pass1 = password1;
  char * pass2 = password2;
  char * pass3 = password3;
  char * pass4 = password4;

  while ( * a == * pass1) {
    if ( * a == '\0') {
      printf("password:%s\n", password1);
      break;
    }
    a++;
    pass1++;
  }
  while ( * b == * pass2) {
    if ( * b == '\0') {
      printf("password:%s\n", password2);
      break;
    }
    b++;
    pass2++;
  }
  while ( * c == * pass3) {
    if ( * c == '\0') {
      printf("password:%s\n", password3);
      break;
    }
    c++;
    pass3++;
  }
  while ( * d == * pass4) {
    if ( * d == '\0') {
      printf("password: %s\n", password4);
      return 1;
    }
    d++;
    pass4++;
  }
  return 0;
}

__global__ void kernel() {
  char i1, i2;

  char password[7];
  password[6] = '\0';

  int i = blockIdx.x + 65;
  int j = threadIdx.x + 65;
  char firstMatch = i;
  char secondMatch = j;

  password[0] = firstMatch;
  password[1] = secondMatch;
  for (i1 = '0'; i1 <= '9'; i1++) {
    for (i2 = '0'; i2 <= '9'; i2++) {
      password[2] = i1;
      password[3] = i2;
      if (is_a_match(password)) {} else {
        //printf("tried: %s\n",password);
      }
    }
  }
}

int time_difference(struct timespec * start, struct timespec * finish, long long int * difference) {
  long long int ds = finish -> tv_sec - start -> tv_sec;
  long long int dn = finish -> tv_nsec - start -> tv_nsec;

  if (dn < 0) {
    ds--;
    dn += 1000000000;
  }
  * difference = ds * 1000000000 + dn;
  return !( * difference > 0);
}

int main() {

  struct timespec start, finish;
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, & start);

  kernel <<< 26, 26 >>> ();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, & finish);
  time_difference( & start, & finish, & time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
    (time_elapsed / 1.0e9));
  return 0;
}