#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables


__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

__host__ __device__ int compareString(char * a, char * b)
{
		return strcmp(a, b);
		// printf("helow\n");
}


__global__ void crack(char * alphabet, char * numbers){

char genRawPass[4];
char * hashed_password = "xtwdwy2271"; // vz03


genRawPass[0] = alphabet[blockIdx.x];
genRawPass[1] = alphabet[blockIdx.y];

genRawPass[2] = numbers[threadIdx.x];
genRawPass[3] = numbers[threadIdx.y];

//firstLetter - 'a' - 'z' (26 characters)
//secondLetter - 'a' - 'z' (26 characters)
//firstNum - '0' - '9' (10 characters)
//secondNum - '0' - '9' (10 characters)

//Idx --> gives current index of the block or thread
// if(compareString(hashed_password , CudaCrypt(genRawPass)) == 0){
// 	printf("Password found: %c%c%c%c = %s\n", genRawPass[0],genRawPass[1],genRawPass[2],genRawPass[3], hashed_password);
// }
// }else{
	// printf("Comparestring : %s\n", CudaCrypt(genRawPass));
// }
// printf("%d", compareString(hashed_password, CudaCrypt(genRawPass)));

// https://forums.developer.nvidia.com/t/cuda-and-char-programming/4824/2


bool match = false;
// char  *result = CudaCrypt(genRawPass);
// printf("%s\n",result);




	 printf("%c %c %c %c = %s\n", genRawPass[0],genRawPass[1],genRawPass[2],genRawPass[3], CudaCrypt(genRawPass));

}

int main(int argc, char ** argv){

char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};

char * gpuAlphabet;
hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

char * gpuNumbers;
hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26); 
hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers );
hipDeviceSynchronize();
return 0;
}













