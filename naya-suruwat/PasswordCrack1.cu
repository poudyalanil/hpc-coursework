#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables


//This function compares the encrypted values obtained from CudaCrypt method with input encrypted value
__device__ int cracker(char* enc,char* gen){
	int flag=0;
	while (*enc != '\0' || *gen != '\0') { 
        if (*enc == *gen) { 
            enc++; 
            gen++; 
        } 
  
        // If two characters are not same 
        // print the difference and exit 
        else if ((*enc == '\0' && *gen != '\0') 
                 || (*enc != '\0' && *gen == '\0') 
                 || *enc != *gen) { 
            flag = 1; 
			//printf("Uequal Strings\n"); 
            break; 
        } 
    } 
  
    // If two strings are exactly same 
    if (flag == 0) { 
		printf("Equal Strings\n"); 
		return 0;
	} 
	else{
		return 1;
	}
}
__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

__global__ void crack(char * alphabet, char * numbers){
int result;
char genRawPass[4];
char encrypted[]= "uqtcvx5144";

genRawPass[0] = alphabet[blockIdx.x];
genRawPass[1] = alphabet[blockIdx.y];

genRawPass[2] = numbers[threadIdx.x];
genRawPass[3] = numbers[threadIdx.y];

char *generated={CudaCrypt(genRawPass)};
result = cracker(encrypted,generated);
int flag=0;
	while (*enc != '\0' || *gen != '\0') { 
        if (*enc == *gen) { 
            enc++; 
            gen++; 
        } 
  
        // If two characters are not same 
        // print the difference and exit 
        else if ((*enc == '\0' && *gen != '\0') 
                 || (*enc != '\0' && *gen == '\0') 
                 || *enc != *gen) { 
            flag = 1; 
			//printf("Uequal Strings\n"); 
            break; 
        } 
    } 
  
    // If two strings are exactly same 
    if (flag == 0) { 
		printf("Equal Strings\n"); 
		return 0;
	} 
	else{
		return 1;
	}


}

int main(int argc, char ** argv){
//char encrypted[]="cxbdwy2734";
char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};

char * gpuAlphabet;
hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

char * gpuNumbers;
hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26); 
hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers);
hipDeviceSynchronize();
return 0;
}