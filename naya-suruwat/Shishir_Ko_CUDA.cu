#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <pthread.h>

pthread_mutex_t mutex = PTHREAD_MUTEX_INITIALIZER;

__device__ int is_a_match (char *attempt) {
	char plain_password1[] = "SH1234";
	char plain_password2[] = "RE2345";
	char plain_password3[] = "EJ3456";
	char plain_password4[] = "AN4567";
	
	char *a = attempt;
	char *b = attempt;
	char *c = attempt;
	char *d = attempt;
	char *p1 = plain_password1;
	char *p2 = plain_password2;
	char *p3 = plain_password3;
	char *p4 = plain_password4;

	while (*a == *p1) {
		if (*a == '\0')
		{
			printf ("Password: %s\n", plain_password1);
			break;
		}
		a++;
		p1++;
	}
	
	while(*b == *p2) {
		if(*b == '\0')
		{
			printf("Password: %s\n",plain_password2);
			break;
		}
		b++;
		p2++;
	}
	
	while(*c == *p3) {
		if(*c == '\0')
		{
			printf("Password: %s\n",plain_password3);
			break;
		}
		c++;
		p3++;
	}
	
	while (*d == *p4) {
		if (*d == '\0')
		{
			printf ("Password: %s\n", plain_password4);
			return 1;
		}
		d++;
		p4++;
	}
	
	return 0;
}

__global__ void kernel () {
	char i1, i2, i3, i4;
	char password [7];
	password [6] = '\0';
	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i;
	char secondMatch = j;
	password [0] = firstMatch;
	password [1] = secondMatch;
	
	for (i1='0'; i1<='9'; i1++) {
		for (i2='0'; i2<='9'; i2++) {
			for (i3='0'; i3<='9'; i3++) {
				for (i4='0'; i4<='9'; i4++) {
					password [2] = i1;
					password [3] = i2;
					password [4] = i3;
					password [5] = i4;
					
					if(is_a_match(password)) {
					}
					else {
						//printf ("tried: %s\n", password);
					}
				}
			}
		}
	}
}

int time_difference (struct timespec *start, struct timespec *finish, long long int *difference) {
	long long int ds = finish->tv_sec - start->tv_sec;
	long long int dn = finish->tv_nsec - start->tv_nsec;
	if (dn < 0) {
		ds--;
		dn += 1000000000;
	}
	*difference = ds * 1000000000 + dn;
	return! (*difference > 0);
}


int main () {
	struct timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);
	pthread_mutex_lock(&mutex);
	kernel <<<26,26>>>();
	hipDeviceSynchronize();
	pthread_mutex_unlock(&mutex);
	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
	return 0;
}

