#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime_api.h>

#include <time.h>

__device__ char* is_a_match(char * attempt) {
  char password1[] = "OKNXRT3171";

  char * newPassword = (char *) malloc(sizeof(char) * 11);

  newPassword[0] = password1[0] - 2;
	newPassword[1] = password1[0] + 2;
	newPassword[2] = password1[0] - 1;
	newPassword[3] = password1[1] - 3;
	newPassword[4] = password1[1] + 3;
	newPassword[5] = password1[1] + 1;
	newPassword[6] = password1[2] - 2;
	newPassword[7] = password1[2] + 2;
	newPassword[8] = password1[3] - 4;
	newPassword[9] = password1[3] + 4;
  newPassword[10] = '\0';

  printf("------");
  // 
  for(int i =0; i<10; i++){
    printf("%s\n", newPassword[i]);
    if(i >= 0 && i < 6){ //checking all lower case letter limits
      printf("%s", newPassword[i]);
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
  // char * a = attempt;
  // char * pass1 = password1;

  // while ( * a == * pass1) {
  //   if ( * a == '\0') {
  //     printf("password:%s\n", password1);
  //     break;
  //   }
  //   a++;
  //   pass1++;
  // }
  // return 0;
  return newPassword;
}

__global__ void kernel() {
  char i1, i2;

  char password[7];
  password[6] = '\0';
  password[0] = blockIdx.x + 65;
  password[1] = threadIdx.x + 65;
  for (i1 = '0'; i1 <= '9'; i1++) {
    for (i2 = '0'; i2 <= '9'; i2++) {
      password[2] = i1;
      password[3] = i2;
      // if (is_a_match(password)) {
      //   printf("%s \n Encrypted\n", is_a_match(password));
      // } else {
      //   //printf("tried: %s\n",password);
      // }
      is_a_match(password);
    }
  }
}

int time_difference(struct timespec * start, struct timespec * finish, long long int * difference) {
  long long int ds = finish -> tv_sec - start -> tv_sec;
  long long int dn = finish -> tv_nsec - start -> tv_nsec;

  if (dn < 0) {
    ds--;
    dn += 1000000000;
  }
  * difference = ds * 1000000000 + dn;
  return !( * difference > 0);
}

int main() {

  struct timespec start, finish;
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, & start);

  kernel <<< 26, 26 >>> ();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, & finish);
  time_difference( & start, & finish, & time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
    (time_elapsed / 1.0e9));
  return 0;
}